#include "hip/hip_runtime.h"
#include "OmnidirectionalCamera.cuh"

#include <opencv2/cudev/ptr2d/glob.hpp>
#include <hip/hip_runtime.h>
#include <>

/*******************************
* cuda_DivAndClip_kernel
*	arguments
* 	src   : input  data pointer (GlobPtrSz)
*	right : output data pointer (GlobPtrSz)
*	left  : output data pointer (GlobPtrSz)
*******************************/
__global__ void cuda_DivAndClip_kernel(const cv::cudev::GlobPtrSz<uchar> src,  cv::cudev::GlobPtrSz<uchar> right ,cv::cudev::GlobPtrSz<uchar> left ,int x_offset ,int y_offset){
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int src_color_tid =  (y+y_offset) * src.step + (3 * (x + x_offset));
    const int right_color_tid = y * right.step + (3 * x);

    int offset = src.cols/2;
    if((x < right.cols) && (y < right.rows)){
	right.data[right_color_tid + 0]  =  src.data[src_color_tid + 0];
	right.data[right_color_tid + 1]  =  src.data[src_color_tid + 1];
	right.data[right_color_tid + 2]  =  src.data[src_color_tid + 2];

	left.data[right_color_tid + 0]  =   src.data[src_color_tid + 0 + offset];
	left.data[right_color_tid + 1]  =   src.data[src_color_tid + 1 + offset];
	left.data[right_color_tid + 2]  =   src.data[src_color_tid + 2 + offset];
   }
}

/*******************************
* cuda_DivAndClip
*	arguments
* 	src   : input  data pointer (GpuMat)
*	right : output data pointer (GpuMat)
*	left  : output data pointer (GpuMat)
*******************************/
void OmnidirectionalCamera::cuda::DivAndClip(cv::cuda::GpuMat &src ,cv::cuda::GpuMat &right, cv::cuda::GpuMat &left, cv::Rect roi){

   int x_offset = roi.x;
   int y_offset = roi.y;
	
	//create image pointer
    cv::cudev::GlobPtrSz<uchar> p_Right = cv::cudev::globPtr(right.ptr<uchar>(), right.step, right.rows, right.cols * right.channels());
    cv::cudev::GlobPtrSz<uchar> p_Left  = cv::cudev::globPtr(left.ptr<uchar>() , left.step , left.rows , left.cols  *  left.channels());
    cv::cudev::GlobPtrSz<uchar> p_Src   = cv::cudev::globPtr(src.ptr<uchar>()  , src.step  , src.rows  , src.cols   * src.channels());

    const dim3 block(32, 24);
    const dim3 grid(cv::cudev::divUp(right.cols, block.x), cv::cudev::divUp(right.rows, block.y));


    cuda_DivAndClip_kernel<<<grid, block>>>( p_Src ,p_Right,p_Left ,x_offset, y_offset);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());

}
